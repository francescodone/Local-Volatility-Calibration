#include "hip/hip_runtime.h"
#include "ProjHelperFun.cu.h"
#include "Constants.h"
#include "TridagPar.cu.h"
#include "CudaKernels.cu.h"


void   run_OrigCPU(  
		 const unsigned int&   outer,
		 const unsigned int&   numX,
		 const unsigned int&   numY,
		 const unsigned int&   numT,
		 const REAL&           s0,
		 const REAL&           t, 
		 const REAL&           alpha, 
		 const REAL&           nu, 
		 const REAL&           beta,
		       REAL*           res   // [outer] RESULT
 ) {

    // calculating cuda dim

    int full_block_size = 256;
    int block_size = 32;

    dim3 block_2(block_size, block_size, 1);

    int  dim_outer = ceil( ((float) outer)/block_size); 
    int  dim_x = ceil( ((float) numX)/block_size );
    int  dim_y = ceil( ((float) numY)/block_size );

    dim3 grid_2 (dim_x, dim_outer, 1);
    dim3 grid_2_2 (dim_y, dim_outer, 1);
    dim3 grid_3 (dim_y, dim_x, outer);
    dim3 grid_3_2 (dim_x, dim_y, outer);

    unsigned int num_blocks_outer = ((outer + (full_block_size - 1)) / full_block_size);

    // ----- ARRAY EXPNASION ------

    PrivGlobs    globs(numX, numY, numT, outer);
    unsigned numZ = max(numX,numY);

    // ----- MAIN LOOP ------


    for( unsigned k = 0; k < outer; ++ k ) {
        initGrid(s0,alpha,nu,t, numX, numY, numT, k, globs);
        initOperator(globs.myX,globs.myDxx, globs.sizeX, k, numX);
        initOperator(globs.myY,globs.myDyy, globs.sizeY, k, numY);
    }

    // allocating memory for setPayoff

    REAL *d_payoff, *d_myX, *d_my_result;
    hipMalloc((void**) &d_payoff, outer * numX * sizeof(REAL));
    hipMalloc((void**) &d_myX, outer * numX * sizeof(REAL));
    hipMalloc((void**) &d_my_result, outer * numX * numY * sizeof(REAL));

    hipMemcpy(d_myX, globs.myX, outer * numX * sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_my_result, globs.myResult, outer * numX * numY * sizeof(REAL), hipMemcpyHostToDevice);

    // ---- setPayoff ----
    
    initPayoff<<<grid_2, block_2>>>(outer, numX, d_payoff, d_myX);
    hipDeviceSynchronize();

    updateGlobsMyResult<<<grid_3, block_2>>>(outer, numX, numY, d_payoff, d_my_result);
    hipDeviceSynchronize();

    hipFree(d_payoff);

     // --- end of setPayoff ----


    REAL *d_myY, *d_myTimeline, *d_myVarX, *d_myVarY, *d_myDxx, *d_myDyy, *d_u, *d_v, *d_dtInv;
    REAL *d_a, *d_b, *d_c, *d_yy, *d_y;
    hipMalloc((void**) &d_myY, outer * numY * sizeof(REAL));
    hipMalloc((void**) &d_myTimeline, outer * numT * sizeof(REAL));
    hipMalloc((void**) &d_myVarX, outer * numX * numY * sizeof(REAL));
    hipMalloc((void**) &d_myVarY, outer * numX * numY * sizeof(REAL));
    hipMalloc((void**) &d_myDxx,    outer * numX *    4 * sizeof(REAL));
    hipMalloc((void**) &d_myDyy,    outer * numY *    4 * sizeof(REAL));
    hipMalloc((void**) &d_u,        outer * numX * numY * sizeof(REAL));
    hipMalloc((void**) &d_v,        outer * numX * numY * sizeof(REAL));
    hipMalloc((void**) &d_dtInv, outer * sizeof(REAL));

    hipMalloc((void**) &d_a, outer * numZ * numZ * sizeof(REAL));
    hipMalloc((void**) &d_b, outer * numZ * numZ * sizeof(REAL));
    hipMalloc((void**) &d_c, outer * numZ * numZ * sizeof(REAL));
    hipMalloc((void**) &d_yy, outer * numX * numY * sizeof(REAL));
    hipMalloc((void**) &d_y, outer * numX * numY * sizeof(REAL));

    hipMemcpy(d_myY, globs.myY, outer * numY * sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_myTimeline, globs.myTimeline, outer * numT * sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_myVarX, globs.myVarX, outer * numX * numY * sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_myVarY, globs.myVarY, outer * numX * numY * sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_myDyy, globs.myDyy, outer * numY * 4 * sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_myDxx,    globs.myDxx,    outer * numX *    4 * sizeof(REAL), hipMemcpyHostToDevice);


    for(int g = globs.sizeT-2;g>=0;--g) { // seq

        // --- updateParams ---      
        updateParams<<<grid_3, block_2>>>(outer, numX, numY, numT, g, alpha,
            beta, nu, d_myX, d_myY, d_myTimeline, d_myVarX, d_myVarY);
        hipDeviceSynchronize();

	    // --- rollback ---
        rollback<<<num_blocks_outer, full_block_size>>>(outer, numT, g, d_myTimeline, d_dtInv);
        hipDeviceSynchronize();   

        // ---- explicit x
        explicitX<<<grid_3, block_2>>>(outer, numX, numY, d_dtInv, d_my_result,
			d_myVarX, d_myDxx, d_u);
        hipDeviceSynchronize();

        // ------ explicit y
        explicitY<<<grid_3_2, block_2>>>(outer,
	        numX, numY, d_my_result, d_myVarY, d_myDyy, d_v, d_u);
        hipDeviceSynchronize();

        // ------- implicit x
        implicitX<<<grid_3_2, block_2>>>(outer, numX, numY, d_dtInv, d_myVarX, d_myDxx, d_a, d_b, d_c);
        hipDeviceSynchronize();

        implicitX_tridag<<<grid_2_2, block_2>>>(outer, numX, numY, d_a, d_b, d_c, d_u, d_yy);
        hipDeviceSynchronize();

        //	------- implicit y
        implicitY_1<<<grid_3, block_2>>>(outer,numX, numY, d_dtInv, d_myVarY, d_myDyy, d_a, d_b, d_c);
        hipDeviceSynchronize();

        implicitY_2<<<grid_3, block_2>>>(outer,numX, numY, d_dtInv, d_u, d_v, d_y);
        hipDeviceSynchronize();

        implicitY_tridag<<<grid_2, block_2>>>(outer,numX, numY, d_a, d_b, 
            d_c, d_y, d_yy, d_my_result);
        hipDeviceSynchronize();
        
    }

    REAL *d_res;
    unsigned *d_myXindex, *d_myYindex;
    hipMalloc((void**) &d_myXindex, outer * sizeof(unsigned));
    hipMalloc((void**) &d_myYindex, outer * sizeof(unsigned));
    hipMalloc((void**) &d_res, outer * sizeof(REAL));

    hipMemcpy(d_myXindex, globs.myXindex, outer * sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy(d_myYindex, globs.myYindex, outer * sizeof(unsigned), hipMemcpyHostToDevice); 

    updateRes<<<num_blocks_outer, full_block_size>>>(outer, numX, numY, d_myXindex, d_myYindex, d_my_result, d_res);
    hipDeviceSynchronize;

    hipMemcpy(res, d_res, outer * sizeof(REAL), hipMemcpyDeviceToHost);

    hipFree(d_myX);
    hipFree(d_myY);
    hipFree(d_myTimeline);
    hipFree(d_myVarX);
    hipFree(d_myVarY);
    hipFree(d_myDxx);
    hipFree(d_myDyy);
    hipFree(d_dtInv);
    hipFree(d_my_result);
    hipFree(d_myXindex);
    hipFree(d_myYindex);
    
    hipFree(d_res);
    
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_yy);
    hipFree(d_y);
}

//#endif // PROJ_CORE_ORIG
