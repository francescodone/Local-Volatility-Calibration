#include "hip/hip_runtime.h"
#include "ProjHelperFun.cu.h"
#include "Constants.h"
#include "TridagPar.cu.h"
#include "CudaKernels.cu.h"


void   run_OrigCPU(  
		 const unsigned int&   outer,
		 const unsigned int&   numX,
		 const unsigned int&   numY,
		 const unsigned int&   numT,
		 const REAL&           s0,
		 const REAL&           t, 
		 const REAL&           alpha, 
		 const REAL&           nu, 
		 const REAL&           beta,
		       REAL*           res   // [outer] RESULT
 ) {

    // calculating cuda dim

    int full_block_size = 256;
    int block_size = 16;

    dim3 block_2(block_size, block_size, 1);

    int  dim_outer = ceil( ((float) outer)/block_size); 
    int  dim_x = ceil( ((float) numX)/block_size );
    int  dim_y = ceil( ((float) numY)/block_size );

    dim3 grid_2 (dim_x, dim_outer, 1);
    dim3 grid_2_2 (dim_y, dim_outer, 1);
    dim3 grid_3 (dim_y, dim_x, outer);
    dim3 grid_3_2 (dim_x, dim_y, outer);

    unsigned int num_blocks_outer = ((outer + (full_block_size - 1)) / full_block_size);

    // ----- ARRAY EXPNASION ------

    // PrivGlobs    globs(numX, numY, numT, outer);
    PrivGlobsCuda    globsCuda(numX, numY, numT, outer);
    unsigned numZ = max(numX,numY);

    // ----- MAIN LOOP ------

    initGridKernel<<<num_blocks_outer, full_block_size>>>(s0,alpha,nu,t, numX, numY, numT, outer, globsCuda);
    hipDeviceSynchronize();

    initOperatorKernel<<<num_blocks_outer, full_block_size>>>(globsCuda.myX,globsCuda.myDxx, globsCuda.sizeX, outer, numX);
    hipDeviceSynchronize();

    initOperatorKernel<<<num_blocks_outer, full_block_size>>>(globsCuda.myY,globsCuda.myDyy, globsCuda.sizeY, outer, numY);
    hipDeviceSynchronize();


    // for( unsigned k = 0; k < outer; ++ k ) {
    //     initGrid(s0,alpha,nu,t, numX, numY, numT, k, globs);
    //     initOperator(globs.myX,globs.myDxx, globs.sizeX, k, numX);
    //     initOperator(globs.myY,globs.myDyy, globs.sizeY, k, numY);
    // }

    // allocating memory for setPayoff

    REAL *d_payoff, *d_myX, *d_my_result;
    hipMalloc((void**) &d_payoff, outer * numX * sizeof(REAL));
    hipMalloc((void**) &d_myX, outer * numX * sizeof(REAL));
    hipMalloc((void**) &d_my_result, outer * numX * numY * sizeof(REAL));

    // hipMemcpy(d_myX, globs.myX, outer * numX * sizeof(REAL), hipMemcpyHostToDevice);
    // hipMemcpy(d_my_result, globs.myResult, outer * numX * numY * sizeof(REAL), hipMemcpyHostToDevice);

    hipMemcpy(d_myX, globsCuda.myX, outer * numX * sizeof(REAL), hipMemcpyDeviceToDevice);
    hipMemcpy(d_my_result, globsCuda.myResult, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToDevice);

    // ---- setPayoff ----
    
    initPayoff<<<grid_2, block_2>>>(outer, numX, d_payoff, d_myX);
    hipDeviceSynchronize();

    updateGlobsMyResult<<<grid_3, block_2>>>(outer, numX, numY, d_payoff, d_my_result);
    hipDeviceSynchronize();

    hipFree(d_payoff);

     // --- end of setPayoff ----


    REAL *d_myY, *d_myTimeline, *d_myVarX, *d_myVarY, *d_myDxx, *d_myDyy, *d_u, *d_v, *d_dtInv;
    REAL *d_a, *d_b, *d_c, *d_yy, *d_y;
    hipMalloc((void**) &d_myY, outer * numY * sizeof(REAL));
    hipMalloc((void**) &d_myTimeline, outer * numT * sizeof(REAL));
    hipMalloc((void**) &d_myVarX, outer * numX * numY * sizeof(REAL));
    hipMalloc((void**) &d_myVarY, outer * numX * numY * sizeof(REAL));
    hipMalloc((void**) &d_myDxx,    outer * numX *    4 * sizeof(REAL));
    hipMalloc((void**) &d_myDyy,    outer * numY *    4 * sizeof(REAL));
    hipMalloc((void**) &d_u,        outer * numX * numY * sizeof(REAL));
    hipMalloc((void**) &d_v,        outer * numX * numY * sizeof(REAL));
    hipMalloc((void**) &d_dtInv, outer * sizeof(REAL));

    hipMalloc((void**) &d_a, outer * numZ * numZ * sizeof(REAL));
    hipMalloc((void**) &d_b, outer * numZ * numZ * sizeof(REAL));
    hipMalloc((void**) &d_c, outer * numZ * numZ * sizeof(REAL));
    hipMalloc((void**) &d_yy, outer * numX * numY * sizeof(REAL));
    hipMalloc((void**) &d_y, outer * numX * numY * sizeof(REAL));

    // hipMemcpy(d_myY, globs.myY, outer * numY * sizeof(REAL), hipMemcpyHostToDevice);
    //hipMemcpy(d_myTimeline, globs.myTimeline, outer * numT * sizeof(REAL), hipMemcpyHostToDevice);
    // hipMemcpy(d_myVarX, globs.myVarX, outer * numX * numY * sizeof(REAL), hipMemcpyHostToDevice);
    // hipMemcpy(d_myVarY, globs.myVarY, outer * numX * numY * sizeof(REAL), hipMemcpyHostToDevice);
    // hipMemcpy(d_myDyy, globs.myDyy, outer * numY * 4 * sizeof(REAL), hipMemcpyHostToDevice);
    // hipMemcpy(d_myDxx,    globs.myDxx,    outer * numX *    4 * sizeof(REAL), hipMemcpyHostToDevice);

    hipMemcpy(d_myY, globsCuda.myY, outer * numY * sizeof(REAL), hipMemcpyDeviceToDevice);
    hipMemcpy(d_myTimeline, globsCuda.myTimeline, outer * numT * sizeof(REAL), hipMemcpyDeviceToDevice);
    hipMemcpy(d_myVarX, globsCuda.myVarX, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToDevice);
    hipMemcpy(d_myVarY, globsCuda.myVarY, outer * numX * numY * sizeof(REAL), hipMemcpyDeviceToDevice);
    hipMemcpy(d_myDyy, globsCuda.myDyy, outer * numY * 4 * sizeof(REAL), hipMemcpyDeviceToDevice);
    hipMemcpy(d_myDxx,    globsCuda.myDxx,    outer * numX *    4 * sizeof(REAL), hipMemcpyDeviceToDevice);

    
    REAL *d_u_T;
    hipMalloc((void**) &d_u_T, outer * numX * numY * sizeof(REAL));


    for(int g = globsCuda.sizeT-2;g>=0;--g) { // seq

        // --- updateParams ---      
        updateParams<<<grid_3, block_2>>>(outer, numX, numY, numT, g, alpha,
            beta, nu, d_myX, d_myY, d_myTimeline, d_myVarX, d_myVarY);
        hipDeviceSynchronize();

	    // --- rollback ---
        rollback<<<num_blocks_outer, full_block_size>>>(outer, numT, g, d_myTimeline, d_dtInv);
        hipDeviceSynchronize();   

        // ---- explicit x

        // matrix transposition 
        
        // matTransposeTiled<<<numX, numY, outer>>>(d_u, d_u_T, numX, numY, outer);
        // hipDeviceSynchronize();


        explicitX<<<grid_3, block_2>>>(outer, numX, numY, d_dtInv, d_my_result,
			d_myVarX, d_myDxx, d_u);
        hipDeviceSynchronize();

        // matTransposeTiled<<<numY, numX, outer>>>(d_u_T, d_u, numY, numX, outer);
        // hipDeviceSynchronize();


        // ------ explicit y
        explicitY<<<grid_3_2, block_2>>>(outer,
	        numX, numY, d_my_result, d_myVarY, d_myDyy, d_v, d_u);
        hipDeviceSynchronize();



        // ------- implicit x
        implicitX<<<grid_3_2, block_2>>>(outer, numX, numY, d_dtInv, d_myVarX, d_myDxx, d_a, d_b, d_c);
        hipDeviceSynchronize();

        implicitX_tridag<<<grid_2_2, block_2>>>(outer, numX, numY, d_a, d_b, d_c, d_u, d_yy);
        hipDeviceSynchronize();

        //	------- implicit y
        implicitY_1<<<grid_3, block_2>>>(outer,numX, numY, d_dtInv, d_myVarY, d_myDyy, d_a, d_b, d_c);
        hipDeviceSynchronize();

        implicitY_2<<<grid_3, block_2>>>(outer,numX, numY, d_dtInv, d_u, d_v, d_y);
        hipDeviceSynchronize();

        implicitY_tridag<<<grid_2, block_2>>>(outer,numX, numY, d_a, d_b, 
            d_c, d_y, d_yy, d_my_result);
        hipDeviceSynchronize();
        
    }

    REAL *d_res;
    unsigned *d_myXindex, *d_myYindex;
    hipMalloc((void**) &d_myXindex, outer * sizeof(unsigned));
    hipMalloc((void**) &d_myYindex, outer * sizeof(unsigned));
    hipMalloc((void**) &d_res, outer * sizeof(REAL));

    hipMemcpy(d_myXindex, globsCuda.myXindex, outer * sizeof(unsigned), hipMemcpyDeviceToDevice);
    hipMemcpy(d_myYindex, globsCuda.myYindex, outer * sizeof(unsigned), hipMemcpyDeviceToDevice); 

    updateRes<<<num_blocks_outer, full_block_size>>>(outer, numX, numY, d_myXindex, d_myYindex, d_my_result, d_res);
    hipDeviceSynchronize();

    hipMemcpy(res, d_res, outer * sizeof(REAL), hipMemcpyDeviceToHost);

    hipFree(d_myX);
    hipFree(d_myY);
    hipFree(d_myTimeline);
    hipFree(d_myVarX);
    hipFree(d_myVarY);
    hipFree(d_myDxx);
    hipFree(d_myDyy);
    hipFree(d_dtInv);
    hipFree(d_my_result);
    hipFree(d_myXindex);
    hipFree(d_myYindex);
    
    hipFree(d_res);
    
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_yy);
    hipFree(d_y);
}

//#endif // PROJ_CORE_ORIG
