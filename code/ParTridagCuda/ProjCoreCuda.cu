#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "TridagKernel.cu.h"
#include "CoreKernel.cu.h"
#include "../OrigImpl/ProjHelperFun.h"
#include "Constants.h"
#include "TridagPar.h"

void updateParams(const unsigned g, const REAL alpha, const REAL beta, const REAL nu, PrivGlobs& globs)
{
#pragma omp parallel for collapse(2) default(shared) schedule(static)
    for(unsigned i=0;i<globs.myX.size();++i)
        for(unsigned j=0;j<globs.myY.size();++j) {
            globs.myVarX[i][j] = exp(2.0*(  beta*log(globs.myX[i])   
                                          + globs.myY[j]             
                                          - 0.5*nu*nu*globs.myTimeline[g] )
                                    );
            globs.myVarY[i][j] = exp(2.0*(  alpha*log(globs.myX[i])   
                                          + globs.myY[j]             
                                          - 0.5*nu*nu*globs.myTimeline[g] )
                                    ); // nu*nu
        }
}

void setPayoff(const REAL strike, PrivGlobs& globs )
{
    vector<REAL> payoffs(globs.myX.size());
    #pragma omp parallel for default(shared) schedule(static)
    for(unsigned i=0;i<globs.myX.size();++i) {
        payoffs[i] = max(globs.myX[i]-strike, (REAL)0.0);
    }

    #pragma omp parallel for collapse(2) default(shared) schedule(static)
    for(unsigned i=0;i<globs.myX.size();++i)
    {
        for(unsigned j=0;j<globs.myY.size();++j)
            globs.myResult[i][j] = payoffs[i];
    }
}

inline void tridag(
    const vector<REAL>&   a,   // size [n]
    const vector<REAL>&   b,   // size [n]
    const vector<REAL>&   c,   // size [n]
    const vector<REAL>&   r,   // size [n]
    const int             n,
          vector<REAL>&   u,   // size [n]
          vector<REAL>&   uu   // size [n] temporary
) {
    int    i, offset;
    REAL   beta;

    u[0]  = r[0];
    uu[0] = b[0];

    for(i=1; i<n; i++) {
        beta  = a[i] / uu[i-1];

        uu[i] = b[i] - beta*c[i-1];
        u[i]  = r[i] - beta*u[i-1];
    }

#if 1
    // X) this is a backward recurrence
    u[n-1] = u[n-1] / uu[n-1];
    for(i=n-2; i>=0; i--) {
        u[i] = (u[i] - c[i]*u[i+1]) / uu[i];
    }
#else
    // Hint: X) can be written smth like (once you make a non-constant)
    for(i=0; i<n; i++) a[i] =  u[n-1-i];
    a[0] = a[0] / uu[n-1];
    for(i=1; i<n; i++) a[i] = (a[i] - c[n-1-i]*a[i-1]) / uu[n-1-i];
    for(i=0; i<n; i++) u[i] = a[n-1-i];
#endif
}


void
rollback( const unsigned g, PrivGlobs& globs ) {
    unsigned numX = globs.myX.size(),
             numY = globs.myY.size();

    unsigned numZ = max(numX,numY);

    unsigned i, j;

    REAL dtInv = 1.0/(globs.myTimeline[g+1]-globs.myTimeline[g]);

    vector<vector<REAL> > u(numY, vector<REAL>(numX));   // [numY][numX]
    vector<vector<REAL> > v(numX, vector<REAL>(numY));   // [numX][numY]

    //	explicit x
    #pragma omp parallel for collapse(2) default(shared) schedule(static)
    for(i=0;i<numX;i++) {
        for(j=0;j<numY;j++) {
            u[j][i] = dtInv*globs.myResult[i][j];

            if(i > 0) { 
              u[j][i] += 0.5*( 0.5*globs.myVarX[i][j]*globs.myDxx[i][0] ) 
                            * globs.myResult[i-1][j];
            }
            u[j][i]  +=  0.5*( 0.5*globs.myVarX[i][j]*globs.myDxx[i][1] )
                            * globs.myResult[i][j];
            if(i < numX-1) {
              u[j][i] += 0.5*( 0.5*globs.myVarX[i][j]*globs.myDxx[i][2] )
                            * globs.myResult[i+1][j];
            }
        }
    }

    //	explicit y
    #pragma omp parallel for collapse(2) default(shared) schedule(static)
    for(j=0;j<numY;j++)
    {
        for(i=0;i<numX;i++) {
            v[i][j] = 0.0;

            if(j > 0) {
              v[i][j] +=  ( 0.5*globs.myVarY[i][j]*globs.myDyy[j][0] )
                         *  globs.myResult[i][j-1];
            }
            v[i][j]  +=   ( 0.5*globs.myVarY[i][j]*globs.myDyy[j][1] )
                         *  globs.myResult[i][j];
            if(j < numY-1) {
              v[i][j] +=  ( 0.5*globs.myVarY[i][j]*globs.myDyy[j][2] )
                         *  globs.myResult[i][j+1];
            }
            u[j][i] += v[i][j]; 
        }
    }

    vector< vector<REAL> > as1(numY, vector<REAL>(numX));
    vector< vector<REAL> > bs1(numY, vector<REAL>(numX));
    vector< vector<REAL> > cs1(numY, vector<REAL>(numX));
    #pragma omp parallel for collapse(2) default(shared) schedule(static)
    for(j=0;j<numY;j++) {
        for(i=0;i<numX;i++) {
            as1[j][i] =		 - 0.5*(0.5*globs.myVarX[i][j]*globs.myDxx[i][0]);
            bs1[j][i] = dtInv - 0.5*(0.5*globs.myVarX[i][j]*globs.myDxx[i][1]);
            cs1[j][i] =		 - 0.5*(0.5*globs.myVarX[i][j]*globs.myDxx[i][2]);            
        }
    }

    //	implicit x
    #pragma omp parallel for default(shared) schedule(static)
    for(j=0;j<numY;j++) {
        // here yy should have size [numX]
        vector<REAL> yy(numX);
        tridagPar(as1[j],bs1[j],cs1[j],u[j],numX,u[j],yy);
    }

    vector< vector<REAL> > as2(numX, vector<REAL>(numY));
    vector< vector<REAL> > bs2(numX, vector<REAL>(numY));
    vector< vector<REAL> > cs2(numX, vector<REAL>(numY));
    vector< vector<REAL> > ys2(numX, vector<REAL>(numY));
    #pragma omp parallel for collapse(2) default(shared) schedule(static)
    for(i=0;i<numX;i++) { 
        for(j=0;j<numY;j++) {  // here a, b, c should have size [numY]
            as2[i][j] =		 - 0.5*(0.5*globs.myVarY[i][j]*globs.myDyy[j][0]);
            bs2[i][j] = dtInv - 0.5*(0.5*globs.myVarY[i][j]*globs.myDyy[j][1]);
            cs2[i][j] =		 - 0.5*(0.5*globs.myVarY[i][j]*globs.myDyy[j][2]);
            ys2[i][j] = dtInv*u[j][i] - 0.5*v[i][j];
        }
    }
    //	implicit y
    #pragma omp parallel for collapse(1) default(shared) schedule(static)
    for(i=0;i<numX;i++) { 
        // here yy should have size [numY]
        vector<REAL> yy(numY);
        tridagPar(as2[i],bs2[i],cs2[i],ys2[i],numY,globs.myResult[i],yy);
    }
}

REAL   value(   PrivGlobs    globs,
                const REAL s0,
                const REAL strike, 
                const REAL t, 
                const REAL alpha, 
                const REAL nu, 
                const REAL beta,
                const unsigned int numX,
                const unsigned int numY,
                const unsigned int numT
) {	
    initGrid(s0,alpha,nu,t, numX, numY, numT, globs);
    initOperator(globs.myX,globs.myDxx);
    initOperator(globs.myY,globs.myDyy);

    setPayoff(strike, globs);
    for(int i = globs.myTimeline.size()-2;i>=0;--i)
    {
        updateParams(i,alpha,beta,nu,globs);
        rollback(i, globs);
    }

    return globs.myResult[globs.myXindex][globs.myYindex];
}

void   run_OrigCPU(  
                const unsigned int&   outer,
                const unsigned int&   numX,
                const unsigned int&   numY,
                const unsigned int&   numT,
                const REAL&           s0,
                const REAL&           t, 
                const REAL&           alpha, 
                const REAL&           nu, 
                const REAL&           beta,
                      REAL*           res   // [outer] RESULT
) {
    #pragma omp parallel for default(shared) schedule(static) if(outer>8)
    for( unsigned i = 0; i < outer; ++ i ) {
        REAL strike;
        PrivGlobs    globs(numX, numY, numT);

        strike = 0.001*i;
        res[i] = value( globs, s0, strike, t,
                        alpha, nu,    beta,
                        numX,  numY,  numT );
    }
}

//#endif // PROJ_CORE_ORIG
